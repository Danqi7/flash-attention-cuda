#include "hip/hip_runtime.h"
#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

#define Br 2
#define Bc 2
#define d 64

#define EPS 0.000001

void cpu_attention(FP *q,FP *k, FP *v, FP *p, FP *o, int n) {
  // Q, K, V, O: [n, d]. usually n>>d
  FP rowMax, rowSum, sVal, oVal;
  int indexq, indexk;
  // S = QK^T; P = softmax(S) row-wise;
  for (int row=0; row<n; row++) {
    rowMax = INFINITY;
    rowSum = 0.0;

    for (int col=0; col<n; col++) {
      sVal = 0;
      indexk = col*d;
      for (indexq = row*d; indexq < (row*d + d); indexq++, indexk++) {
        sVal += q[indexq] * k[indexk];
      }
      rowMax = max(rowMax, sVal);
      p[row*n+col] = sVal; // unnormalized, wt sub, wt exponentials
    }

    // Subtract max of row for numerical stability
    // Sum up elements in the same row
    for (int col=0; col < n; col++) {
      rowSum += exp(p[row*n+col] - rowMax);
    }

    // Normalize for each row
    for (int col=0; col < n; col++)
      p[row*n+col] = exp(p[row*n+col] - rowMax) / rowSum;
    
    // Check rowProb sums up to 1
    FP rowProb = 0.;
    for (int col=0; col < n; col++)
      rowProb += p[row*n+col];
    if (rowProb - 1.0 > EPS)
      printf("[CPU func]rowProb is not 1! rowProb: %.3f\n", rowProb);
      
    // O = PV for each row
    for (int v_col = 0; v_col < d; v_col++) {
      oVal = 0;
      for (int col=0; col < n; col++) {
        oVal += p[row*n+col] * v[col*d+v_col];
      }
      o[row*d+v_col] -= oVal; // NOTE: This calculates the diff between CPU and GPU computations.
    }
    
  }
}

// Fused flash attention kernel
__global__ void flash_attn(FP *Q, FP *K, FP *V, FP *O, FP *l, FP *m, int n) {
    /*
        Q, V, K, O : [n, d]; l, m : [n,]
        Divide Q into Q1, Q2, ..., QTr of shape [Br, d]
        Divide K, V in K1/V1, ..., KTc/VTc of shape [Bc, d]
        Divide O into O1, O2, ..., OTr of shape [Br, d]
        Divide l, m into l1/m1, ..., lTr/mTr of shape [Br,]
    */

  //int Tr = (n + Br - 1) / Br;
  int Tc = (n + Bc - 1) / Bc;
  
  __shared__ FP Kj[Bc][d];
  __shared__ FP Vj[Bc][d];
  __shared__ FP Qi[Br][d];
  __shared__ FP Oi[Br][d];
  __shared__ FP Sij[Br][Bc];
  
  __shared__ FP tmp_Oi[Br][d];
  __shared__ FP Pij_delta[Br][Bc];

  __shared__ FP li[Br];
  __shared__ FP mi[Br];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int col = tx + blockDim.x * blockIdx.x; // col is tx when GridDim.x = 1
  int row = ty + blockDim.y * blockIdx.y;
  if (row == 0 & col == 0){
    printf("starting Tc loop...\n");
  }
  for (int j = 0; j < Tc; j++) {
    // Load Kj, Vj from HBM to on-chip SRAM
    if ((j*Bc+tx) < n) {
      for (int idx=0; idx < d; idx++) {
          //Kj[tx][idx] = K[col][idx]
          Kj[tx][idx] = K[j*Bc*d + tx*d + idx];
          Vj[tx][idx] = V[j*Bc*d + tx*d + idx];
      }
    }
    
    // Load Qi, Oi, li, mi from HBM to on-chip SRAM
    if (row < n) {
        for (int idx=0; idx < d; idx++) {
            //Q[ty][idx] = Q[row*d][idx]
            Qi[ty][idx] = Q[row * d + idx];
            Oi[ty][idx] = O[row * d + idx];
        }
        li[ty] = l[row];
        mi[ty] = m[row];
    }

    // make sure Qi, Oi, Kj, Vj are loaded
    __syncthreads();

    // Compute Sij = QiKj^T
    FP sVal = 0.0;
    for (int idx=0; idx < d; idx++) {
        sVal += Qi[ty][idx] * Kj[tx][idx];
    }
    Sij[ty][tx] = sVal; // Sij is of size [Br x Bc]
    // make sure Sij is computed
    __syncthreads();

    // Compute rowmax m^~_ij
    // mij_delta[ty] = max(mij_delta[ty], Sij[ty*Bc + tx]); // Assume mij_delta is initialized to -inf
    // __syncthreads();
    FP mij_delta = -INFINITY;
    for (int idx=0; idx < Bc; idx++) {
        mij_delta = max(mij_delta, Sij[ty][idx]);
    }
    __syncthreads();

    // P^~_ij
    //Pij_delta = exp(S[ty*Bc + tx] - m_delta[ty]);
    Pij_delta[ty][tx] = exp(Sij[ty][tx] - mij_delta);
    __syncthreads();

    // l^~_ij
    //lij_delta[ty] += Pij_delta; // Assume l_delta is initialized to 0
    FP lij_delta = 0;
    for (int idx=0; idx < Bc; idx++) {
        lij_delta += Pij_delta[ty][idx];
    }
    __syncthreads();

    // Compute new m, l
    FP mi_new = max(mi[ty], mij_delta);
    FP coeffold = exp(mi[ty] - mi_new);
    FP coeffcur = exp(mij_delta - mi_new);
    FP li_new = coeffold * li[ty] + coeffcur * lij_delta; // per row in Br

    // Update Oi: each row doing ([1,Bc]).dot([Bc,d]) = [1,d]
    for (int idx=0; idx < d; idx++) {
        tmp_Oi[ty][idx] = 0;
    }
    for (int idx=0; idx < d; idx++) {
        //tmp_Oi[ty*d + idx] += (li[ty] * coeffold * Oi[ty*d+idx] + coeffcur * Pij_delta * Vj[tx*d+idx]) / li_new;
        //race condition?
        tmp_Oi[ty][idx] += coeffcur * Pij_delta[ty][tx] * Vj[tx][idx];
    }
    if (tx == 0) {
      for (int idx=0; idx < d; idx++) {
        tmp_Oi[ty][idx] += li[ty] * coeffold * Oi[ty][idx];
      }
      for (int idx=0; idx < d; idx++) {
        tmp_Oi[ty][idx] = tmp_Oi[ty][idx] / li_new;
      }
    }
    __syncthreads();

    // Update Oi in HBM
    if (row < n) {
        for (int idx=0; idx < d; idx++) {
            O[row*d + idx] = tmp_Oi[ty][idx];
        }
    }

    // Update li, mi in HBM
    l[row] = li_new;
    m[row] = mi_new;
    __syncthreads();

  }
}


int main(int argc, char *argv[]) {
  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_X = 1; //Grid x dimension
  int Grid_Dim_Y = 1; // Grid y dim
  int Block_Dim_X = 1; //Block dimension
  int Block_Dim_Y = 1;

  int n;
  FP *q,*k,*v, *o; // Q,K,V,O are (n, d) inputs, S,P are intermediate (n, n) matrix
  FP *l, *m;
  FP *dev_q, *dev_k, *dev_v, *dev_o;
  FP *dev_l, *dev_m;
  size_t Qsize; // number of bytes in arrays
  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc!=2) {
    printf("Usage: flash_attn <matrix dim n>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  //d = atoi(argv[2]);
  //Block_Dim_Y = atoi(argv[3]); // Non-square block, [Br, Bc]
  //Block_Dim_X = atoi(argv[4]);
  Block_Dim_Y = Br; // Non-square block, [Br, Bc]
  Block_Dim_X = Bc;
  if (Block_Dim_Y*Block_Dim_X > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  hipSetDevice(gpunum);
  printf("Debugging....\n");
  printf("Using device %d\n",gpunum);
  printf("Matrix Q,K,V Dimension = [%d, %d]\n", n, d);
  Qsize = n * d * sizeof(FP);
  //Psize = n * n * sizeof(FP);
  //printf("malloc ...\n");
  q = (FP*) malloc(Qsize); // dynamically allocated memory for arrays on host
  k = (FP*) malloc(Qsize);
  v = (FP*) malloc(Qsize);
  o = (FP*) malloc(Qsize); // final output
  l = (FP*) malloc(n*sizeof(FP));
  m = (FP*) malloc(n*sizeof(FP));

  srand(12345);
  for(i=0;i < n;i++)
    for(j=0;j < d;j++) {
      //printf("q: %d, %d:\n", i, j);
      q[i * d + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }
  for(i=0;i < n;i++)
    for(j=0;j < d;j++) {
      //printf("k: %d, %d:\n", i, j);
      k[i * d + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }
  for(i=0;i < n;i++)
    for(j=0;j < d;j++) {
      //printf("v: %d, %d:\n", i, j);
      v[i * d + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }
  printf("Init O, m, l...");
  // Init O, m, l
  for (i=0;i < n;i++)
    for(j=0;j < d;j++) {
      //printf("o: %d, %d:\n", i, j);
      o[i*d+j] = 0.0;
    }

  for (i=0;i<n;i++) {
    //printf("l/m: %d:\n", i);
    l[i] = 0;
    m[i] = -INFINITY;
  }
  printf("START GPU CODE...");
  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_q, Qsize); // allocate memory on device
  hipMalloc((void**)&dev_k, Qsize);
  hipMalloc((void**)&dev_v, Qsize);
  hipMalloc((void**)&dev_o, Qsize);
  hipMalloc((void**)&dev_l, n * sizeof(FP));
  hipMalloc((void**)&dev_m, n * sizeof(FP));

  hipMemcpy(dev_q, q , Qsize ,hipMemcpyHostToDevice);
  hipMemcpy(dev_k, k , Qsize ,hipMemcpyHostToDevice);
  hipMemcpy(dev_v, v , Qsize ,hipMemcpyHostToDevice);
  hipMemcpy(dev_o, o , Qsize ,hipMemcpyHostToDevice);
  hipMemcpy(dev_l, l , n * sizeof(FP) ,hipMemcpyHostToDevice);
  hipMemcpy(dev_m, m , n * sizeof(FP) ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // hipEventSynchronize(start); // not needed

  Grid_Dim_X = 1; //  how many blocks in X direction
  Grid_Dim_Y = (n + Block_Dim_Y - 1) / Block_Dim_Y; // how many blocks in Y direction
  int Tc = (n + Bc - 1) / Bc;
  if (Grid_Dim_X*Block_Dim_X*Tc < n) {
    printf("Error, number of threads in x dimensions less than number of array elements\n");
    exit (-1);
  }
  if (Grid_Dim_Y*Block_Dim_Y < n) {
    printf("QK^T: Error, number of threads in y dimensions less than number of array elements\n");
    exit (-1);
  }
  printf("Block_Dim = (%d, %d), Grid_Dim = (%d, %d)\n", Block_Dim_X, Block_Dim_Y, Grid_Dim_X, Grid_Dim_Y);
  dim3 Grid(Grid_Dim_X, Grid_Dim_Y); //Grid structure
  dim3 Block(Block_Dim_X, Block_Dim_Y); //Block structure
  
  flash_attn<<<Grid,Block>>>(dev_q, dev_k, dev_v, dev_o, dev_l, dev_m, n);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipMemcpy(o, dev_o, Qsize ,hipMemcpyDeviceToHost);

  printf("Time for Flash Attention on GPU: %f ms.\n", elapsed_time_ms);


// START OF OPTIONAL SECTION THAT CAN BE OMITTED

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // hipEventSynchronize(start); // not needed

  //cpu_matrixmult(a,b,c, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

  FP* p = (FP*) malloc(n*n*sizeof(FP));
  cpu_attention(q,k,v, p,o, n);

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double oerror, sumv, vi;
  sumv = 0.;
  for(i=0;i < n*d; i++) {
    vi = (double) v[i];
    sumv += vi * vi;
  }
  sumv = sqrt(sumv);
  oerror =  sumv / (n*d);
  printf("Approximate relative error between GPU and CPU: %e\n", oerror);
  free(p);
  
// END OF OPTIONAL SECTION THAT CAN BE OMITTED
// -------------- clean up ---------------------------------------

  free(q);
  free(k);
  free(v);
  free(o);
  free(l);
  free(m);
  hipFree(dev_q);
  hipFree(dev_k);
  hipFree(dev_v);
  hipFree(dev_o);
  hipFree(dev_l);
  hipFree(dev_m);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
